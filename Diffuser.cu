#include "hip/hip_runtime.h"
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//        This file is part of the Spatiocyte package
//
//        Copyright (C) 2006-2009 Keio University
//        Copyright (C) 2010-2014 RIKEN
//
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//
// Spatiocyte is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public
// License as published by the Free Software Foundation; either
// version 2 of the License, or (at your option) any later version.
// 
// Spatiocyte is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
// See the GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public
// License along with Spatiocyte -- see the file COPYING.
// If not, write to the Free Software Foundation, Inc.,
// 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
// 
//END_HEADER
//
// written by Satya Arjunan <satya.arjunan@gmail.com>
//

#include <time.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/system/cuda/detail/bulk/bulk.hpp>
#include <hiprand/hiprand_kernel.h>
#include <Diffuser.hpp>
#include <Compartment.hpp>
#include <Model.hpp>
#include <Reaction.hpp>
#include <random>

Diffuser::Diffuser(const double D, Species& species):
  D_(D),
  species_(species),
  compartment_(species_.get_compartment()),
  mols_(species_.get_mols()),
  voxels_(species_.get_compartment().get_lattice().get_voxels()),
  offsets_(species_.get_compartment().get_offsets()),
  species_id_(species_.get_id()),
  vac_id_(species_.get_vac_id()),
  seed_(0) {
}

void Diffuser::initialize() {
  Model& model(species_.get_model());
  stride_ = model.get_stride();
  id_stride_ = species_id_*stride_;
  is_reactive_.resize(model.get_species().size(), false);
  reactions_.resize(model.get_species().size(), NULL);
  substrate_mols_.resize(model.get_species().size(), NULL);
  product_mols_.resize(model.get_species().size(), NULL);
  reacteds_.resize(mols_.size()+1, 0);
 
  std::vector<Reaction*>& reactions(species_.get_reactions());
  for(unsigned i(0); i != reactions.size(); ++i) {
    std::vector<Species*>& substrates(reactions[i]->get_substrates());
    for(unsigned j(0); j != substrates.size(); ++j) {
      voxel_t reactant_id(substrates[j]->get_id());
      if(reactant_id != species_id_) {
        reactions_[reactant_id] = reactions[i];
        is_reactive_[reactant_id] = true;
        substrate_mols_[reactant_id] = thrust::raw_pointer_cast(substrates[j]->get_mols().data());
        product_mols_[reactant_id] = thrust::raw_pointer_cast(reactions[i]->get_products()[0]->get_mols().data());
      } 
    } 
  } 
  /*
  std::cout << "My name:" << species_.get_name_id() << std::endl;
  for(unsigned i(0); i != is_reactive_.size(); ++i) {
    std::cout << "\t" << is_reactive_[i] << " reactant name:" << model.get_species()[i]->get_name_id() << std::endl;
    std::cout << "\t" << (reactions_[i] != NULL) << std::endl;
  }
  */
}

double Diffuser::get_D() const {
  return D_;
}

struct generate {
  __host__ __device__ generate(
      const unsigned mol_size,
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const mol_t* offsets,
      voxel_t* voxels):
    mol_size_(mol_size),
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    offsets_(offsets),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    thrust::default_random_engine rng;
    rng.discard(seed_+index);
    thrust::uniform_int_distribution<unsigned> u(0, 11);
    const unsigned rand(u(rng));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    //Atomically put the current molecule id, index+id_stride_ at the target
    //voxel if it is vacant: 
    const voxel_t tar_mol_id(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, finalize walk:
    if(tar_mol_id == vac_id_) {
      voxels_[vdx] = vac_id_;
      return val;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned mol_size_;
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const mol_t* offsets_;
  voxel_t* voxels_;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        size,
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}


/* Verified random walk without any reaction checks: 41.9 s
struct generate {
  __host__ __device__ generate(
      const unsigned mol_size,
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const mol_t* offsets,
      voxel_t* voxels):
    mol_size_(mol_size),
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    offsets_(offsets),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    thrust::default_random_engine rng;
    rng.discard(seed_+index);
    thrust::uniform_int_distribution<unsigned> u(0, 11);
    const unsigned rand(u(rng));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    //Atomically put the current molecule id, index+id_stride_ at the target
    //voxel if it is vacant: 
    const voxel_t tar_mol_id(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, finalize walk:
    if(tar_mol_id == vac_id_) {
      voxels_[vdx] = vac_id_;
      return val;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned mol_size_;
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const mol_t* offsets_;
  voxel_t* voxels_;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        size,
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}
*/

/* Verified random walk without reaction: 42.0 s
struct generate {
  __host__ __device__ generate(
      const unsigned mol_size,
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const bool* is_reactive,
      const mol_t* offsets,
      umol_t* reacteds,
      voxel_t* voxels):
    mol_size_(mol_size),
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    is_reactive_(is_reactive),
    offsets_(offsets),
    reacteds_(reacteds),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    thrust::default_random_engine rng;
    rng.discard(seed_+index);
    thrust::uniform_int_distribution<unsigned> u(0, 11);
    const unsigned rand(u(rng));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    //Atomically put the current molecule id, index+id_stride_ at the target
    //voxel if it is vacant: 
    const voxel_t tar_mol_id(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, finalize walk:
    if(tar_mol_id == vac_id_) {
      voxels_[vdx] = vac_id_;
      return val;
    }
    //it is occupied, so check if it is reactive and add reacted:
    const voxel_t tar_id(tar_mol_id/stride_);
    if(is_reactive_[tar_id]) {
      //const unsigned old(atomicAdd(reacteds_+mol_size_, 1));
      reacteds_[index] = tar_mol_id;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned mol_size_;
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const bool* is_reactive_;
  const mol_t* offsets_;
  umol_t* reacteds_;
  voxel_t* voxels_;
};

struct is_reacted {
  __device__ bool operator()(const umol_t reacted) {
    return reacted;
  }
};

struct react {
  __host__ __device__ react(
      const unsigned mol_size,
      umol_t* reacteds):
    mol_size_(mol_size),
    reacteds_(reacteds) {}
  __device__ umol_t operator()(const unsigned index, const umol_t mol) const {
      const unsigned old(atomicSub(reacteds_+mol_size_, 1));
    reacteds_[index] = 0;
    return mol;
  }
  const unsigned mol_size_;
  umol_t* reacteds_;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  reacteds_.resize(size+1);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        size,
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&is_reactive_[0]),
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&reacteds_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}
*/

/*
struct generate {
  __host__ __device__ generate(
      const unsigned mol_size,
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const bool* is_reactive,
      const mol_t* offsets,
      umol_t* reacteds,
      voxel_t* voxels):
    mol_size_(mol_size),
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    is_reactive_(is_reactive),
    offsets_(offsets),
    reacteds_(reacteds),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    thrust::default_random_engine rng;
    rng.discard(seed_+index);
    thrust::uniform_int_distribution<unsigned> u(0, 11);
    const unsigned rand(u(rng));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    //Atomically put the current molecule id, index+id_stride_ at the target
    //voxel if it is vacant: 
    const voxel_t tar_mol_id(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, finalize walk:
    if(tar_mol_id == vac_id_) {
      voxels_[vdx] = vac_id_;
      return val;
    }
    //it is occupied, so check if it is reactive and add reacted:
    const voxel_t tar_id(tar_mol_id/stride_);
    if(is_reactive_[tar_id]) {
      //const unsigned old(atomicAdd(reacteds_+mol_size_, 1));
      reacteds_[index] = tar_mol_id;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned mol_size_;
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const bool* is_reactive_;
  const mol_t* offsets_;
  umol_t* reacteds_;
  voxel_t* voxels_;
};

struct is_reacted {
  __device__ bool operator()(const umol_t reacted) {
    return reacted;
  }
};

struct react {
  __host__ __device__ react(
      const unsigned mol_size,
      umol_t* reacteds):
    mol_size_(mol_size),
    reacteds_(reacteds) {}
  __device__ umol_t operator()(const unsigned index, const umol_t mol) const {
      const unsigned old(atomicSub(reacteds_+mol_size_, 1));
    reacteds_[index] = 0;
    return mol;
  }
  const unsigned mol_size_;
  umol_t* reacteds_;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  reacteds_.resize(size+1);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        size,
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&is_reactive_[0]),
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&reacteds_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  thrust::transform_if(thrust::device,
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      reacteds_.begin(),
      mols_.begin(),
      react(
        size,
        thrust::raw_pointer_cast(&reacteds_[0])),
      is_reacted());
  seed_ += size;
}
*/

/* Verified random walk with thrust::random : 44.1 s
struct generate {
  __host__ __device__ generate(
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const bool* is_reactive,
      const mol_t* offsets,
      umol_t* reacteds,
      voxel_t* voxels):
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    is_reactive_(is_reactive),
    offsets_(offsets),
    reacteds_(reacteds),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    thrust::default_random_engine rng;
    rng.discard(seed_+index);
    thrust::uniform_int_distribution<unsigned> u(0, 11);
    const unsigned rand(u(rng));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    const voxel_t res(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, walk:
    if(res == vac_id_) {
      voxels_[vdx] = vac_id_;
      reacteds_[index] = 0;
      return val;
    }
    //If occupied, check and add reacted:
    const voxel_t tar_id(res/stride_);
    if(is_reactive_[tar_id]) {
      reacteds_[index] = res;
    }
    else {
      reacteds_[index] = 0;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const bool* is_reactive_;
  const mol_t* offsets_;
  umol_t* reacteds_;
  voxel_t* voxels_;
};

struct is_reacted {
  __device__ bool operator()(const umol_t reacted) {
    return reacted;
  }
};

struct react {
  __device__ umol_t operator()(const umol_t mol, const umol_t reacted) const {
    return mol;
  }
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  reacteds_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&is_reactive_[0]),
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&reacteds_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  thrust::transform_if(thrust::device,
      mols_.begin(),
      mols_.end(),
      reacteds_.begin(),
      reacteds_.begin(),
      mols_.begin(),
      react(),
      is_reacted());
  seed_ += size;
}
*/

/*
// With pseudo reaction and non-overlap population: 43.2 s
struct is_reacted {
  __device__ bool operator()(const umol_t reacted) {
    return reacted;
  }
};

struct react {
  __device__ umol_t operator()(const umol_t mol, const umol_t reacted) const {
    return mol;
  }
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  reacteds_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&is_reactive_[0]),
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&reacteds_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  thrust::transform_if(thrust::device,
      mols_.begin(),
      mols_.end(),
      reacteds_.begin(),
      reacteds_.begin(),
      mols_.begin(),
      react(),
      is_reacted());
  seed_ += size;
}
*/

/*
//With reaction check list: 41.5 s
struct generate {
  __host__ __device__ generate(
      const unsigned seed,
      const voxel_t stride,
      const voxel_t id_stride,
      const voxel_t vac_id,
      const bool* is_reactive,
      const mol_t* offsets,
      umol_t* reacteds,
      voxel_t* voxels):
    seed_(seed),
    stride_(stride),
    id_stride_(id_stride),
    vac_id_(vac_id),
    is_reactive_(is_reactive),
    offsets_(offsets),
    reacteds_(reacteds),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned index, const umol_t vdx) const {
    hiprandState s;
    hiprand_init(seed_+index, 0, 0, &s);
    float ranf(hiprand_uniform(&s)*11.999999);
    const unsigned rand((unsigned)truncf(ranf));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets_[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    const voxel_t res(atomicCAS(voxels_+val, vac_id_, index+id_stride_));
    //If not occupied, walk:
    if(res == vac_id_) {
      voxels_[vdx] = vac_id_;
      reacteds_[index] = 0;
      return val;
    }
    //If occupied, check and add reacted:
    const voxel_t tar_id(res/stride_);
    if(is_reactive_[tar_id]) {
      reacteds_[index] = res;
    }
    else {
      reacteds_[index] = 0;
    }
    //Stay at original position:
    return vdx;
  }
  const unsigned seed_;
  const voxel_t stride_;
  const voxel_t id_stride_;
  const voxel_t vac_id_;
  const bool* is_reactive_;
  const mol_t* offsets_;
  umol_t* reacteds_;
  voxel_t* voxels_;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  reacteds_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(size),
      mols_.begin(),
      mols_.begin(),
      generate(
        seed_,
        stride_,
        id_stride_,
        vac_id_,
        thrust::raw_pointer_cast(&is_reactive_[0]),
        thrust::raw_pointer_cast(&offsets_[0]),
        thrust::raw_pointer_cast(&reacteds_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}
*/

/*
//Use atomicCAS to avoid race condition: 39.1 s
struct generate {
  __host__ __device__ generate(const mol_t* _offsets, voxel_t* _voxels):
    offsets(_offsets), voxels(_voxels) {} 
  __device__ umol_t operator()(const unsigned n, const umol_t vdx) const {
    hiprandState s;
    hiprand_init(n, 0, 0, &s);
    float ranf(hiprand_uniform(&s)*11.999999);
    const unsigned rand((unsigned)truncf(ranf));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    const voxel_t res(atomicCAS(voxels+val, 0, 1));
    if(res == 0) {
      voxels[vdx] = 0;
      return val;
    }
    return vdx;
  }
  const mol_t* offsets;
  voxel_t* voxels;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      mols_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0]), thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}
*/

/*
//Perform walk in predicate, only single transformation function: 38.7 s
struct generate {
  __host__ __device__ generate(const mol_t* _offsets, voxel_t* _voxels):
    offsets(_offsets), voxels(_voxels) {} 
  __device__ umol_t operator()(const unsigned n, const umol_t vdx) const {
    hiprandState s;
    hiprand_init(n, 0, 0, &s);
    float ranf(hiprand_uniform(&s)*11.999999);
    const unsigned rand((unsigned)truncf(ranf));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    if(val < 0 || val > NUM_VOXEL) {
      val = vdx;
    }
    if(!voxels[val]) {
      voxels[val] = true;
      voxels[vdx] = false;
      return val;
    }
    else {
      return vdx;
    }
  }
  const mol_t* offsets;
  voxel_t* voxels;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      mols_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0]), thrust::raw_pointer_cast(&voxels_[0])));
  seed_ += size;
}
*/

/*
//Simplified full collision with transform_if: very small performance
//improvement
struct generate {
  __host__ __device__ generate(const mol_t* _offsets):
    offsets(_offsets) {} 
  __device__ umol_t operator()(const unsigned n, const umol_t vdx) const {
    hiprandState s;
    hiprand_init(n, 0, 0, &s);
    float ranf(hiprand_uniform(&s)*11.999999);
    const unsigned rand((unsigned)truncf(ranf));
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets[rand+(24&(-odd_lay))+(12&(-odd_col))]);
    if(val < 0 || val > NUM_VOXEL) {
      return vdx;
    }
    return val;
  }
  const mol_t* offsets;
};

struct is_occupied {
  __device__ bool operator()(const voxel_t voxel) {
    return ((bool)voxel);
  }
};

struct update {
  __device__ umol_t operator()(const umol_t mol) const {
    return mol;
  }
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  tars_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> stencil(voxels_.begin(),
        tars_.begin());
  thrust::transform_if(
      mols_.begin(),
      mols_.end(),
      stencil,
      tars_.begin(),
      update(),
      is_occupied());
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> vacants(voxels_.begin(),
        mols_.begin());
  thrust::fill_n(thrust::device, vacants, size, false);
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> occupieds(voxels_.begin(),
        tars_.begin());
  thrust::fill_n(thrust::device, occupieds, size, true);
  thrust::copy(tars_.begin(), tars_.end(), mols_.begin());
  seed_ += size;
}
*/


/*
//Full collision lattice with transform_if: same performance
struct is_vacant {
  __device__ bool operator()(const voxel_t voxel) {
    return (bool)voxel;
  }
};

struct update {
  __device__ umol_t operator()(const voxel_t tar, const umol_t mol) const {
    return mol;
  }
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  tars_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> stencil(lattice_.begin(),
        tars_.begin());
  thrust::transform_if(thrust::device, 
      tars_.begin(),
      tars_.end(),
      mols_.begin(),
      stencil,
      tars_.begin(),
      update(),
      is_vacant());
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> vacants(lattice_.begin(),
        mols_.begin());
  thrust::fill_n(thrust::device, vacants, size, true);
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> occupieds(lattice_.begin(),
        tars_.begin());
  thrust::fill_n(thrust::device, occupieds, size, false);
  thrust::copy(tars_.begin(), tars_.end(), mols_.begin());
  //thrust::copy(mols_.begin(), mols_.end(), box_mols_[0].begin());
  seed_ += size;
}
*/


/*
//Full collision check with lattice: 0.3 s (1,600,000 molecules, 42.5 s)
struct update {
  __host__ __device__ update(const voxel_t* _lattice):
    lattice(_lattice) {} 
  __device__ umol_t operator()(const umol_t tar, const umol_t mol) const {
    if(lattice[tar]) {
      return tar;
    }
    return mol;
  }
  const voxel_t* lattice;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  tars_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::transform(thrust::device, 
      tars_.begin(),
      tars_.end(),
      mols_.begin(),
      tars_.begin(),
      update(thrust::raw_pointer_cast(&lattice_[0])));
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> vacants(lattice_.begin(),
        mols_.begin());
  thrust::fill_n(thrust::device, vacants, mols_.size(), true);
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> occupieds(lattice_.begin(),
        tars_.begin());
  thrust::fill_n(thrust::device, occupieds, tars_.size(), false);
  thrust::copy(tars_.begin(), tars_.end(), mols_.begin());
  //thrust::copy(mols_.begin(), mols_.end(), box_mols_[0].begin());
  seed_ += mols_.size();
}
*/

/*
//Partial collision check with lattice: 0.2 s
struct update {
  __host__ __device__ update(bool* _lattice):
    lattice(_lattice) {} 
  __device__ umol_t operator()(const umol_t tar, const umol_t mol) const {
    if(lattice[tar]) {
      lattice[tar] = false;
      lattice[mol] = true;
      return tar;
    }
    return mol;
  }
  bool* lattice;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  tars_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::transform(thrust::device, 
      tars_.begin(),
      tars_.end(),
      mols_.begin(),
      mols_.begin(),
      update(thrust::raw_pointer_cast(&lattice_[0])));
  //thrust::copy(mols_.begin(), mols_.end(), box_mols_[0].begin());
  seed_ += mols_.size();
}
*/

/*
//Full collision check using molecule list only: 4.3s

struct update {
  __host__ __device__ update(const size_t _size, const umol_t* _mols):
    size(_size), mols(_mols) {} 
  __device__ umol_t operator()(const umol_t tar, const umol_t mol) const {
    for(unsigned i(0); i != size; ++i) {
      if(tar == mols[i]) {
        return mol;
      }
    }
    return tar;
  }
  const size_t size;
  const umol_t* mols;
};

void Diffuser::walk() {
  const size_t size(mols_.size());
  tars_.resize(size);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+size),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::transform(thrust::device, 
      tars_.begin(),
      tars_.end(),
      mols_.begin(),
      tars_.begin(),
      update(size, thrust::raw_pointer_cast(&mols_[0])));
  thrust::copy(tars_.begin(), tars_.end(), mols_.begin());
  //thrust::copy(mols_.begin(), mols_.end(), box_mols_[0].begin());
  seed_ += mols_.size();
}
*/


/*
struct generate {
  __host__ __device__ generate(mol_t* _offsets):
    offsets(_offsets) {} 
  __device__ umol_t operator()(const unsigned n, const umol_t vdx) const {
    thrust::default_random_engine rng(hash(n));
    thrust::uniform_int_distribution<int> uniform(0, 11);
    const bool odd_lay((vdx/NUM_COLROW)&1);
    const bool odd_col((vdx%NUM_COLROW/NUM_ROW)&1);
    mol2_t val(mol2_t(vdx)+offsets[uniform(rng)+(24&(-odd_lay))+
        (12&(-odd_col))]);
    if(val < 0 || val > NUM_VOXEL) {
      return vdx;
    }
    return val;
  }
  mol_t* offsets;
};
*/


/*
//Collisions not check, with intersection: 7s
void Diffuser::walk() {
  tars_.resize(mols_.size());
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(seed_),
      thrust::counting_iterator<unsigned>(seed_+mols_.size()),
      mols_.begin(),
      tars_.begin(),
      generate(thrust::raw_pointer_cast(&offsets_[0])));
  thrust::sort(thrust::device, tars_.begin(), tars_.end());
  collisions_.resize(mols_.size());
  thrust::set_intersection(thrust::device, mols_.begin(), mols_.end(),
      tars_.begin(), tars_.end(), collisions_.begin());
  //if(!collisions.size()) { 
    thrust::copy(tars_.begin(), tars_.end(), mols_.begin());
  //}
  //thrust::copy(mols_.begin(), mols_.end(), box_mols_[0].begin());
  seed_ += mols_.size();
}
*/

/*
//Sequential original 10.5 s
void Diffuser::walk(umol_t* mols, const unsigned size) {
  for (unsigned i(0); i != size; ++i) {
    umol_t tar(compartment_.get_tar(mols[i], rng_.Ran16_12()));
    for(unsigned j(0); j != size; ++j) {
      if(mols[j] == tar) {
        goto next;
      }
    }
    mols[i] = tar;
next:
    continue;
  }
}
*/
