#include "hip/hip_runtime.h"
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//        This file is part of the Spatiocyte package
//
//        Copyright (C) 2006-2009 Keio University
//        Copyright (C) 2010-2014 RIKEN
//
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//
// Spatiocyte is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public
// License as published by the Free Software Foundation; either
// version 2 of the License, or (at your option) any later version.
// 
// Spatiocyte is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
// See the GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public
// License along with Spatiocyte -- see the file COPYING.
// If not, write to the Free Software Foundation, Inc.,
// 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
// 
//END_HEADER
//
// written by Satya Arjunan <satya.arjunan@gmail.com>
//

#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/execution_policy.h>
#include <Common.hpp>
#include <Random.hpp>

Random::Random(std::mt19937::result_type seed):
  engine_(seed) {
}

unsigned Random::ran(const unsigned min, const unsigned max)
{
  return std::uniform_int_distribution<unsigned>{min, max}(engine_);
}

struct generate {
  __host__ __device__ generate(const unsigned _a, const unsigned _b):
    a(_a), b(_b) {;} 
  __device__ float operator()(const unsigned n) const {
    hiprandState s;
    hiprand_init(n, 0, 0, &s);
    float ranf(hiprand_uniform(&s));
    ranf *= (b - a + 0.999999);
    ranf += a;
    return (unsigned)truncf(ranf);
  }
  unsigned a, b;
};

RandomGPU::RandomGPU(const unsigned min, const unsigned max,
    const unsigned size, const unsigned seed):
  max_(max),
  min_(min),
  size_(size),
  cnt_(0),
  seed_(seed),
  data_(size),
  hdata_(size) {
  initialize();
}

void RandomGPU::initialize() { 
  cnt_ = 0;
  thrust::counting_iterator<unsigned> begin(seed_);
  thrust::transform(thrust::device, begin, begin+size_, data_.begin(), 
                    generate(min_, max_));
  thrust::copy(data_.begin(), data_.end(), hdata_.begin());
  seed_ += size_;
}

umol_t RandomGPU::ran() {
  if(cnt_ >= size_) {
    initialize();
  }
  return hdata_[cnt_++];
}
