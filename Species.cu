#include "hip/hip_runtime.h"
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//        This file is part of the Spatiocyte package
//
//        Copyright (C) 2006-2009 Keio University
//        Copyright (C) 2010-2013 RIKEN
//
//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
//
//
// Spatiocyte is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public
// License as published by the Free Software Foundation; either
// version 2 of the License, or (at your option) any later version.
// 
// Spatiocyte is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
// See the GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public
// License along with Spatiocyte -- see the file COPYING.
// If not, write to the Free Software Foundation, Inc.,
// 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
// 
//END_HEADER
//
// written by Satya Arjunan <satya.arjunan@gmail.com>
//

#include <iostream>
#include <algorithm>
#include <string>
#include <time.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <Species.hpp>
#include <Compartment.hpp>
#include <Reaction.hpp>
#include <Model.hpp>

Species::Species(const std::string name, const unsigned nmols, const double D,
    Model& model, Compartment& compartment, Species& vacant,
    const bool is_structure_species):
  compartment_(compartment),
  model_(model),
  vacant_(vacant),
  voxels_(compartment_.get_lattice().get_voxels()),
  name_(get_init_name(name)),
  init_nmols_(nmols),
  is_structure_species_(is_structure_species),
  id_(model.push_species(*this)),
  vac_id_(vacant_.get_id()),
  diffuser_(D, *this) {
}

void Species::initialize() {
  if(mols_.size() < 80000) {
    mols_.reserve(10000);
  }
  diffuser_.initialize();
}

void Species::push_reaction(Reaction& reaction) {
  if(std::find(reactions_.begin(), reactions_.end(), &reaction) ==
      reactions_.end()) { 
    reactions_.push_back(&reaction);
  }
}

std::vector<Reaction*>& Species::get_reactions() {
  return reactions_;
}

/*
__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

struct populate_lattice {
  __host__ __device__ populate_lattice(const voxel_t _stride_id, const umol_t _size, 
      voxel_t* _voxels):
    stride_id(_stride_id),
    size(_size),
    voxels(_voxels) {} 
  __device__ umol_t operator()(const unsigned n) const {
    unsigned int seed = hash(n);
    thrust::default_random_engine rng(seed);
    thrust::uniform_int_distribution<unsigned> u(0, size);
    unsigned rand(u(rng));
    while(voxels[rand]) {
      rand = u(rng);
    }
    voxels[rand] = stride_id+n;
    return rand;
  }
  const voxel_t stride_id;
  const umol_t size;
  voxel_t* voxels;
};
*/

/*
struct populate_lattice {
  __host__ __device__ populate_lattice(
      const unsigned max_threads,
      const voxel_t stride_id,
      const umol_t _size, 
      voxel_t* _voxels):
    stride_id(_stride_id),
    size(_size),
    mols(_voxels),
    voxels(_voxels) {} 
  __device__ umol_t operator()(const unsigned n) const {
    thrust::default_random_engine rng;
    rng.discard(n);
    thrust::uniform_int_distribution<unsigned> u(0, size);
    unsigned rand(u(rng));
    while(voxels[rand]) {
      rand = u(rng);
    }
    voxels[rand] = stride_id+n;
    return rand;
  }
  const unsigned max_threads_;
  const voxel_t stride_id_;
  const umol_t mols_size_;
  const umol_t voxels_size_;
  voxel_t* voxels_;
  umol_t* mols_;
};



void Species::populate() { 
  unsigned dev(0);
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  unsigned max_threads(deviceProp.maxThreadsPerMultiProcessor);
  unsigned cnts(std::min(max_threads, init_mols_));
  mols_.resize(init_nmols_);
  thrust::for_each(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(cnts),
      populate_lattice(
        max_threads,
        get_id()*model_.get_stride(),
        mols_.size(),
        voxels_.size(),
        thrust::raw_pointer_cast(&mols_[0]),
        thrust::raw_pointer_cast(&voxels_[0])));
}
*/


struct populate_lattice {
  __host__ __device__ populate_lattice(
      const unsigned seed,
      const umol_t mol_size,
      const voxel_t vac_id,
      const voxel_t stride_id,
      const umol_t voxel_size, 
      voxel_t* voxels):
    seed_(seed),
    mol_size_(mol_size),
    vac_id_(vac_id),
    stride_id_(stride_id),
    voxel_size_(voxel_size),
    voxels_(voxels) {} 
  __device__ umol_t operator()(const unsigned n) const {
    thrust::default_random_engine rng(seed_);
    rng.discard(n);
    thrust::uniform_int_distribution<unsigned> u(0, voxel_size_);
    unsigned rand(u(rng));
    voxel_t res(atomicCAS(voxels_+rand, vac_id_, stride_id_+n));
    unsigned cnt(0);
    while(res != vac_id_) {
      rng.discard(n+mol_size_*(++cnt));
      rand = u(rng);
      res = atomicCAS(voxels_+rand, vac_id_, stride_id_+n);
    }
    return rand;
  }
  const unsigned seed_;
  const umol_t mol_size_;
  const voxel_t vac_id_;
  const voxel_t stride_id_;
  const umol_t voxel_size_;
  voxel_t* voxels_;
};



void Species::populate() {
  const unsigned seed(id_);
  mols_.resize(init_nmols_);
  thrust::transform(thrust::device, 
      thrust::counting_iterator<unsigned>(0),
      thrust::counting_iterator<unsigned>(init_nmols_),
      mols_.begin(),
      populate_lattice(
        seed,
        init_nmols_,
        vac_id_,
        get_id()*model_.get_stride(),
        voxels_.size(),
        thrust::raw_pointer_cast(&voxels_[0])));
}

void Species::populate_in_lattice() {
  mols_.resize(host_mols_.size());
  thrust::copy(host_mols_.begin(), host_mols_.end(), mols_.begin());
  thrust::permutation_iterator<thrust::device_vector<voxel_t>::iterator,
    thrust::device_vector<umol_t>::iterator> population(
        voxels_.begin(), mols_.begin());
  thrust::fill_n(thrust::device, population, mols_.size(), get_id());
  if(diffuser_.get_D()) {
    host_mols_.clear();
    std::vector<umol_t>().swap(host_mols_);
  } else {
    mols_.clear();
    thrust::device_vector<umol_t>().swap(mols_);
  }
}

void Species::push_host_mol(const umol_t vdx) {
  host_mols_.push_back(vdx);
}

bool Species::is_structure_species() const {
  return is_structure_species_;
}

bool Species::is_root_structure_species() const {
  return (this == &vacant_);
}

Model& Species::get_model() const {
  return model_;
}

voxel_t Species::get_id() const {
  return id_;
}

voxel_t Species::get_vac_id() const {
  return vac_id_;
}

Diffuser& Species::get_diffuser() {
  return diffuser_;
}

Compartment& Species::get_compartment() {
  return compartment_;
}

Species& Species::get_vacant() {
  return vacant_;
}

const std::string& Species::get_name() const {
  return name_;
}

const std::string Species::get_name_id() const {
  /*
  std::stringstream sid;
  sid << (unsigned)get_id();
  return std::string(get_name()+":"+sid.str());
  */
  return std::string(get_name()+":id:"+std::to_string(get_id())); // c++11
}

const std::string Species::get_init_name(const std::string name) const {
  if(is_root_structure_species()) {
    return std::string(compartment_.get_name()+"/"+name);
  }
  return std::string(vacant_.get_name()+"/"+name);
}

std::vector<umol_t>& Species::get_host_mols() {
  if(diffuser_.get_D()) {
    host_mols_.resize(mols_.size());
    thrust::copy(mols_.begin(), mols_.end(), host_mols_.begin());
  }
  return host_mols_;
}

thrust::device_vector<umol_t>& Species::get_mols() {
  return mols_;
}
